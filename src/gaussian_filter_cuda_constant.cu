#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

#include "gaussian_filter.h"
#include "gaussian_processing.h"
#include "gaussian_filter_cuda.h"

// Constant memory declaration (must be global scope)
#define MAX_KERNEL_SIZE 128
__constant__ float const_kernel[MAX_KERNEL_SIZE];

__device__ void convolve_pixel_horizontal_constant( unsigned char* image, int x, int y, int width, int height, 
                                            int kernel_size, float* out_r, float* out_g, float* out_b) {

    int range = kernel_size / 2;
    *out_r = *out_g = *out_b = 0.0f;
    
    for (int k = -range; k <= range; ++k) {
        int neighbor_x = x + k; // horizontal movement
        if (neighbor_x >= 0 && neighbor_x < width) {
            int base_idx = (y * width + neighbor_x) * 4;
            float weight = const_kernel[k + range]; // read from constant memory

            *out_r += image[base_idx + 0] * weight;
            *out_g += image[base_idx + 1] * weight;
            *out_b += image[base_idx + 2] * weight;
        }
    }
}

__device__ void convolve_pixel_vertical_constant( unsigned char* image, int x, int y, int width, int height, 
                                            int kernel_size, float* out_r, float* out_g, float* out_b) {

    int range = kernel_size / 2;
    *out_r = *out_g = *out_b = 0.0f;
    
    for (int k = -range; k <= range; ++k) {
        int neighbor_y = y + k; // vertical movement
        if (neighbor_y >= 0 && neighbor_y < height) {
            int base_idx = (neighbor_y * width + x) * 4;
            float weight = const_kernel[k + range]; // read from constant memory

            *out_r += image[base_idx + 0] * weight;
            *out_g += image[base_idx + 1] * weight;
            *out_b += image[base_idx + 2] * weight;
        }
    }
}

// This is the kernel (runs on all SMs)
// direction: 0 = horizontal, 1 = vertical
__global__ void gaussian_filter_cuda_convolve_constant( unsigned char* dev_in, unsigned char* dev_out, int width, int height, 
                                                        int kernel_size, int direction)  {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // returns global index position (i.e. block position + thread position)
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float r, g, b;

        if (direction == 0){
            convolve_pixel_horizontal_constant(dev_in, x, y, width, height, kernel_size, &r, &g, &b);
        } else {
            convolve_pixel_vertical_constant(dev_in, x, y, width, height, kernel_size, &r, &g, &b);
        }

        int out_idx = (y * width + x) * 4;
        dev_out[out_idx + 0] = (uint8_t)r;
        dev_out[out_idx + 1] = (uint8_t)g;
        dev_out[out_idx + 2] = (uint8_t)b;
        dev_out[out_idx + 3] = dev_in[out_idx + 3]; // copy the alpha channel directly from input to output buffer
    }
}

// Helper function to allocate GPU memory
hipError_t allocate_device_memory(unsigned char** dev_in, unsigned char** dev_temp, unsigned char** dev_out, // double pointer because we are modifying caller's pointer
                                    size_t image_size) {
    hipError_t err;

    // Allocate dev_in
    err = hipMalloc((void**)dev_in, image_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_in failed: %s\n", hipGetErrorString(err));
        return err;  // Return error, let caller handle cleanup
    }

    // Allocate dev_temp
    err = hipMalloc((void**)dev_temp, image_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_temp failed: %s\n", hipGetErrorString(err));
        return err;  // Caller will clean up dev_in
    }

    // Allocate dev_out
    err = hipMalloc((void**)dev_out, image_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_out failed: %s\n", hipGetErrorString(err));
        return err;  // Caller will clean up dev_in and dev_temp
    }

    return hipSuccess;  // All allocations succeeded
}

// Helper function to copy image from host (CPU) to device (GPU) memory
hipError_t copy_to_device(unsigned char* dev_in, unsigned char* image, float* kernel, size_t image_size, int kernel_size) {
    hipError_t err;

    // Copy image to device
    err = hipMemcpy(dev_in, image, image_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy H2D image failed: %s\n", hipGetErrorString(err));
        return err;
    }

    // Copy kernel to constant memory
    err = hipMemcpyToSymbol(HIP_SYMBOL(
        const_kernel),                    // Destination: __constant__ symbol
        kernel,                          // Source: host memory
        kernel_size * sizeof(float),
        0,
        hipMemcpyHostToDevice
    );
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(err));
        return err;
    }

    return hipSuccess;
}

// Helper fn to launch kernels
hipError_t launch_convolution_kernels(unsigned char* dev_in, unsigned char* dev_temp, unsigned char* dev_out,
                                        int width, int height, int kernel_size, dim3 gridSize, dim3 blockSize) {
    hipError_t err;

    // Horizontal pass
    gaussian_filter_cuda_convolve_constant<<<gridSize, blockSize>>>(
        dev_in, dev_temp, width, height, kernel_size, 0  // direction=0 (horizontal)
    );
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Horizontal kernel launch failed: %s\n", hipGetErrorString(err));
        return err;
    }

    // Vertical pass
    gaussian_filter_cuda_convolve_constant<<<gridSize, blockSize>>>(
        dev_temp, dev_out, width, height, kernel_size, 1  // direction=1 (vertical)
    );
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Vertical kernel launch failed: %s\n", hipGetErrorString(err));
        return err;
    }

    // Wait for completion
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel synchronization failed: %s\n", hipGetErrorString(err));
        return err;
    }

    return hipSuccess;
}

// Helper to copy image back to host
hipError_t copy_to_host(unsigned char* host_image, unsigned char* dev_out, size_t image_size) {
    hipError_t err;

    err = hipMemcpy(host_image, dev_out, image_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy D2H failed: %s\n", hipGetErrorString(err));
        return err;
    }

    return hipSuccess;
}


// Host function
  extern "C" __host__ void gaussian_filter_cuda_constant(unsigned char* image, int width, int height, float sigma, int kernel_size) {
      // Validate inputs
      if (!image) {
          fprintf(stderr, "CUDA: NULL image pointer\n");
          return;
      }
      if (width <= 0 || height <= 0 || kernel_size <= 0) {
          fprintf(stderr, "CUDA: Invalid dimensions w=%d h=%d k=%d\n", width, height, kernel_size);
          return;
      }

      // Precompute Gaussian kernel on CPU
      float* kernel = precompute_gaussian_kernel(kernel_size, sigma);
      if (!kernel) return;

      // Calculate sizes
      size_t image_size = width * height * CHANNELS_PER_PIXEL * sizeof(unsigned char);

      // Configure launch parameters
      dim3 blockSize(16, 16);
      dim3 gridSize((width + 15) / 16, (height + 15) / 16);

      // Initialize device pointers to NULL (important!)
      // hipFree(NULL) is safe and does nothing
      unsigned char *dev_in = NULL;
      unsigned char *dev_temp = NULL;
      unsigned char *dev_out = NULL;

      hipError_t err;

      // Allocate device memory
      err = allocate_device_memory(&dev_in, &dev_temp, &dev_out, image_size);
      if (err != hipSuccess) {
          goto cleanup;  // Jump to cleanup (same function, so this works!)
      }
      // Copy image + kernel data to device
      err = copy_to_device(dev_in, image, kernel, image_size, kernel_size);
      if (err != hipSuccess) {
          goto cleanup;
      }
      // Launch convolutional kernels
      err = launch_convolution_kernels(dev_in, dev_temp, dev_out, width, height, kernel_size, gridSize, blockSize);
      if (err != hipSuccess) {
          goto cleanup;
      }
      // Copy result back to host
      err = copy_to_host(image, dev_out, image_size);
      if (err != hipSuccess) {
          goto cleanup;
      }

  cleanup:
      // Free device memory
      if (dev_in)   hipFree(dev_in);
      if (dev_temp) hipFree(dev_temp);
      if (dev_out)  hipFree(dev_out);

      // Free host memory
      free(kernel);
  }