#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

#include "gaussian_filter.h"
#include "gaussian_processing.h"


// Warmup GPU so 1st run isn't super slow
__global__ void warmup_kernel() {
      // Empty kernel that does minimal work
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      if (idx == 0) {
          // Minimal operation to ensure kernel actually runs
          int dummy = 1 + 1;
      }
  }

// C wrapper function to launch warmup kernel
extern "C" void warmup_gpu(void) {
    warmup_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

// Device helper functions
__device__ void convolve_pixel_horizontal(
    unsigned char* image,
    int x, int y, int width, int height,
    const float* kernel, int kernel_size,
    float* out_r, float* out_g, float* out_b
) {
    int range = kernel_size / 2;
    *out_r = *out_g = *out_b = 0.0f;

    for (int k = -range; k <= range; ++k) {
        int neighbor_x = x + k;
        if (neighbor_x >= 0 && neighbor_x < width) {
            int base_idx = (y * width + neighbor_x) * 4;
            float weight = kernel[k+range];

            *out_r += image[base_idx + 0] * weight;
            *out_g += image[base_idx + 1] * weight;
            *out_b += image[base_idx + 2] * weight;
        }
    }
}

// Vertical convolution helper
__device__ void convolve_pixel_vertical(
    unsigned char* image,
    int x, int y, int width, int height,
    const float* kernel, int kernel_size,
    float* out_r, float* out_g, float* out_b
) {
    int range = kernel_size / 2;
    *out_r = *out_g = *out_b = 0.0f;

    for (int k = -range; k <= range; ++k) {
        int neighbor_y = y + k;
        if (neighbor_y >= 0 && neighbor_y < height) {
            int base_idx = (neighbor_y * width + x) * 4;
            float weight = kernel[k + range];

            *out_r += image[base_idx + 0] * weight;
            *out_g += image[base_idx + 1] * weight;
            *out_b += image[base_idx + 2] * weight;
        }
    }
}

// This is the kernel (runs on all SMs)
__global__ void gaussian_filter_cuda_convolve(
    unsigned char* dev_in, unsigned char* dev_out,
    int width, int height, const float* kernel, int kernel_size, int direction // 0 = horizontal, 1 = vertical
) {
        int x = blockIdx.x * blockDim.x + threadIdx.x; // returns global index position (i.e. block position + thread position)
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < width && y < height) {
            float r, g, b;

            if (direction == 0){
                convolve_pixel_horizontal(dev_in, x, y, width, height, kernel, kernel_size, &r, &g, &b);
            } else {
                convolve_pixel_vertical(dev_in, x, y, width, height, kernel, kernel_size, &r, &g, &b);
            }

            int out_idx = (y * width + x) * 4;
            dev_out[out_idx + 0] = (uint8_t)r;
            dev_out[out_idx + 1] = (uint8_t)g;
            dev_out[out_idx + 2] = (uint8_t)b;
            dev_out[out_idx + 3] = dev_in[out_idx + 3]; // copy the alpha channel directly from input to output buffer
        }
    }


// Entry point
extern "C" __host__ void gaussian_filter_cuda(unsigned char* image, int width, int height, float sigma, int kernel_size) {
    
    // Validate input parameters
    if (!image) {
        fprintf(stderr, "CUDA: NULL image pointer\n");
        return;
    }
    if (width <= 0 || height <= 0 || kernel_size <= 0) {
        fprintf(stderr, "CUDA: Invalid dimensions w=%d h=%d k=%d\n", width, height, kernel_size);
        return;
    }
    

    // Precompute the 1D Gaussian kernel
    float* kernel = precompute_gaussian_kernel(kernel_size, sigma);
    if (!kernel) return;

    // Declare dimensions
    size_t image_size = width * height * CHANNELS_PER_PIXEL * sizeof(unsigned char);
    size_t kernel_size_bytes = kernel_size * sizeof(float);

    dim3 blockSize(16, 16); // Create 2D block of size 16x16 -> 256 threads (8 warps) -> easily fits in 4070
    dim3 gridSize((width + 15) / 16, (height + 15) / 16); // Calculates grid size (i.e. number of blocks to cover image)
    // +15 ensures coverage of partial blocks

    // Allocate GPU memory
    unsigned char *dev_in, *dev_temp, *dev_out;
    float *dev_kernel;

    hipError_t err;
    err = hipMalloc((void**)&dev_in, image_size); 
    // double pointer because we are modifying the pointer itself (write a GPU memory address into pointer variable), not it's data
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_in failed: %s\n", hipGetErrorString(err));
        free(kernel); return;
    }

    err = hipMalloc((void**)&dev_temp, image_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_temp failed: %s\n", hipGetErrorString(err));
        hipFree(dev_in); free(kernel); return;
    }

    err = hipMalloc((void**)&dev_out, image_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_out failed: %s\n", hipGetErrorString(err));
        hipFree(dev_in); hipFree(dev_temp); free(kernel); return;
    }

    err = hipMalloc((void**)&dev_kernel, kernel_size_bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc dev_kernel failed: %s\n", hipGetErrorString(err));
        hipFree(dev_in); hipFree(dev_temp); hipFree(dev_out); free(kernel); return;
    }


    // Transfer original image from system memory to VRAM
      err = hipMemcpy(dev_in, image, image_size, hipMemcpyHostToDevice);
      if (err != hipSuccess) {
          fprintf(stderr, "hipMemcpy H2D image failed: %s\n", hipGetErrorString(err));
          goto cleanup;
      }

      err = hipMemcpy(dev_kernel, kernel, kernel_size_bytes, hipMemcpyHostToDevice);
      if (err != hipSuccess) {
          fprintf(stderr, "hipMemcpy H2D kernel failed: %s\n", hipGetErrorString(err));
          goto cleanup;
      }

    // LAUNCH THE KERNELS

    // Horizontal CUDA Pass
    gaussian_filter_cuda_convolve<<<gridSize, blockSize>>>(
        dev_in, dev_temp, width, height, dev_kernel, kernel_size, 0  // Original width/height
    );
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Horizontal kernel launch failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Vertical CUDA Pass
    gaussian_filter_cuda_convolve<<<gridSize, blockSize>>>(
        dev_temp, dev_out, width, height, dev_kernel, kernel_size, 1
    );
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Vertical kernel launch failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Wait for kernels to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel synchronization failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Copy convolved result from VRAM back to system memory
    err = hipMemcpy(image, dev_out, image_size, hipMemcpyDeviceToHost);
    if(err != hipSuccess) {
        fprintf(stderr, "hipMemcpy D2H (final) failed: %s\n", hipGetErrorString(err));
    }

    cleanup:
        hipFree(dev_in);
        hipFree(dev_temp);
        hipFree(dev_out);
        hipFree(dev_kernel);
        free(kernel);
}